
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}

__global__ void kernel(int *a, int N) {
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  a[i]=i;
}

int main() {
  
  int N = 4097;
  int threads = 128;
  int blocks = (N+threads-1)/threads ;
  int *a;

  printf("No blocks: %d\n", blocks);
  
  gpuErrchk(hipMallocManaged(&a, N*sizeof(int)));
    
  kernel<<<blocks, threads>>>(a, N);
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipPeekAtLastError());

  for(int i=0;i<10;i++) {
    printf("%d\n",a[i]);
  }

  hipFree(a);
  return 0;
}
